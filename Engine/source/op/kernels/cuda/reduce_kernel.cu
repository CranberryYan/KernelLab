#include "hip/hip_runtime.h"
// v0带宽利用率: 62.67%
// v0内存吞吐量: 144.33GB/s
// v1: 解决 warp divergent
// v1带宽利用率: 86.46%
// v1内存吞吐量: 199.35GB/s
// v2: 解决 bank conflict
// v2带宽利用率: 89.95%
// v2内存吞吐量: 207.42GB/s
// v3: 展开最后一个warp的循环
// v3带宽利用率: 92.26%
// v4内存吞吐量: 674.85GB/s(比较接近760GB/s的理论值)


#include <hipcub/hipcub.hpp>
#include "tensor/tensor.h"
#include "reduce_kernel.cuh"
#include "base/cuda_config.h"

namespace kernel {
__global__ void reduce_kernel_v0(const float* input, float* output) {
  // 256个thread
  // 256 * 32/8 = 1024Bytes -> 1kb
  // 3080: 单个SM的L1 cache 128kb
  extern __shared__ float smem[];
  uint32_t tid = threadIdx.x;
  uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  // gmem -> smem
  smem[tid] = input[gid];
  __syncthreads();

  // [1, 2, 3, 4, 5, 6, 7, 8]
  // [1 + 2, 2, 3 + 4, 4, 5 + 6, 6, 7 + 8, 8]
  // [1 + 2 + 3 + 4, 2, 3 + 4, 5 + 6 + 7 + 8, 6, 7 + 8, 8]
  // [1 + 2 + 3 + 4 + 5 + 6 + 7 + 8, 2, 3 + 4, 5 + 6 + 7 + 8, 6, 7 + 8, 8]
  for (uint32_t i = 1; i < blockDim.x; i *= 2) {
    if (tid % (2 * i) == 0) {
      smem[tid] += smem[tid + i];
    }
    __syncthreads();
  }

  if (tid == 0) {
    output[blockIdx.x] = smem[0];
  }
}

__global__ void reduce_kernel_v1(const float* input, float* output) {
  extern __shared__ float smem[];
  uint32_t tid = threadIdx.x;
  uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  // gmem -> smem
  smem[tid] = input[gid];
  __syncthreads();

  // 针对v0
  //  在0号warp中, 有一半会进入if, 另一半不会进入 -> divergent
  //  [1, 2, 3, 4, 5, 6, 7, 8]
  //  [1 + 5, 2 + 6, 3 + 7, 4 + 8, 5, 6, 7, 8]
  //  [1 + 5 + 3 + 7, 2 + 6 + 4 + 8, 3 + 7, 4 + 8, 5, 6, 7, 8]
  //  [1 + 5 + 3 + 7 + 2 + 6 + 4 + 8, 2 + 6 + 4 + 8, 3 + 7, 4 + 8, 5, 6, 7, 8]
  // 修改后
  //  前一半的warp会进入if, 后一半的warp不会进入, 直到最后一次, 0号warp会divergent
  for (uint32_t i = 1; i < blockDim.x; i *= 2) {
    uint32_t index = tid * i * 2;
    if (index < blockDim.x) {
      smem[index] += smem[index + i];
    }
    __syncthreads();
  }

  if (tid == 0) {
    output[blockIdx.x] = smem[0];
  }
}

__global__ void reduce_kernel_v2(const float* input, float* output) {
  extern __shared__ float smem[];
  uint32_t tid = threadIdx.x;
  uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  // gmem -> smem
  smem[tid] = input[gid];
  __syncthreads();

  // 针对v1
  //  i = 1
  //  0号warp的0号thread: smem[0] += smem[1]
  //  0号warp的16号thread: smem[32] += smem[33]
  //  此时会conflict smem[0] 和 smem[32] 会 conflict

  // eg: thread_num = 128
  // 0号warp不会conflict
  //  0 += 65
  //  31 += 96(也是bank0, 为什么不算conflict)
  //  虽然也是访问bank0, 但是和0号warp不是一个warp,不同warp之间不会在同一个时钟周期, 不同warp之间不会bank conflict

  // [1, 2, 3, 4, 5, 6, 7, 8]
  // [1 + 5, 2 + 6, 3 + 7, 4 + 8, 5, 6, 7 ,8]
  // [1 + 5 + 2 + 6, 2 + 6, 3 + 7 + 4 + 8, 4 + 8, 5, 6, 7, 8]
  // [1 + 5 + 2 + 6 + 3 + 7 + 4 + 8, ...]
  for (uint32_t i = blockDim.x / 2; i > 0; i >>= 1) {
    if (tid < i) {
      smem[tid] += smem[tid + i];
    }
  __syncthreads();
  }

  if (tid == 0) {
    output[blockIdx.x] = smem[0];
  }
}

// volatile: 告诉编译器 “不要优化这段内存访问”，每次都要真的访问内存
//  为了确保: 每次读取 smem[tid + x] 都是真的从共享内存中读取，而不是从寄存器中读取可能旧的值
__device__ void unroll_last_warp(volatile float *smem, uint32_t tid) {
  smem[tid] += smem[tid + 32];
  smem[tid] += smem[tid + 16];
  smem[tid] += smem[tid + 8];
  smem[tid] += smem[tid + 4];
  smem[tid] += smem[tid + 2];
  smem[tid] += smem[tid + 1];
}

__global__ void reduce_kernel_v3(const float* input, float* output) {
  uint32_t tid = threadIdx.x;
  uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  // gmem -> smem
  extern __shared__ float smem[]; // 外部已经定义smem的大小(注意要使用extern关键字)
  smem[tid] = input[gid];
  __syncthreads();

  // [1, 2, 3, 4, 5, 6, 7, 8]
  // [1 + 5, 2 + 6, 3 + 7, 4 + 6, 5, 6, 7, 8]
  for (int i = blockDim.x / 2; i > 32; i >>= 1) {
    if (tid < i) {
      smem[tid] += smem[tid + i];
    }
    __syncthreads();
  }

  // unroll last warp
  if (tid < 32) {
    unroll_last_warp(smem, tid);
  }

  // 每个block的第0个thread, 将结果smem -> gmem
  if (tid == 0) {
    output[blockIdx.x] = smem[0];
  }
  __syncthreads();
}

void reduce_kernel_cu(const tensor::Tensor &input,
                      tensor::Tensor &output,
                      para::reduce_para para,
                      void* stream) {
  CHECK_EQ(input.is_empty(), false);
  CHECK_EQ(output.is_empty(), false);

  int32_t thread_num = para.thread_num;
  int32_t block_num = para.block_num;

  dim3 grid(block_num);
  dim3 block(thread_num);

  if (stream) {
    hipStream_t stream_ = static_cast<ihipStream_t*>(stream);
    reduce_kernel_v3<<<grid, block, thread_num * sizeof(float), stream_>>>(
      input.ptr<float>(), output.ptr<float>());
  } else {
    reduce_kernel_v3<<<grid, block, thread_num * sizeof(float)>>>(
      input.ptr<float>(), output.ptr<float>());
  }
}
} // namespace kernel
