#include "hip/hip_runtime.h"
#include "rope_kernel.cuh"

namespace kernel {
__global__ void sin_cos_calc(int head_size, int max_seq_len,
                             float* sin_cache, float* cos_cache) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int head_dim = gid % head_size;
  for (int pos = 0; pos < max_seq_len; ++pos) {
    float freq = 1.0f / pow
      (10000.0f, static_cast<float>(head_dim) / static_cast<float>(head_size));
    float val = static_cast<float>(pos) * freq;
    float fsi = sinf(val);
    float fco = cosf(val);
    *(sin_cache + pos * head_size + head_dim) = fsi;
    *(cos_cache + pos * head_size + head_dim) = fco;
  }
}

__device__ void rope_calc(float fco, float fsi, float* vec, int32_t idx) {
  float2* vec_ptr = reinterpret_cast<float2*>(vec + idx);
  float2 vec_value = *vec_ptr;
  *vec_ptr =
    make_float2(vec_value.x * fco - vec_value.y * fsi,
                vec_value.x * fsi + vec_value.y * fco);
}

__global__ void rope_kernel_cu_fp32(int pos, int dim, int kv_dim, int head_size,
                                    const float* input_q,
                                    const float* input_k,
                                    const float* sin_cache,
                                    const float* cos_cache) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int idx = gid * 2;
  if (idx >= dim) {
    return;
  }

  int head_dim = idx % head_size;
  float fsi = *(sin_cache + pos * head_size + head_dim);
  float fco = *(cos_cache + pos * head_size + head_dim);

  rope_calc(fco, fsi, const_cast<float*>(input_q), idx);
  if (idx >= kv_dim) {
    return;
  }
  rope_calc(fco, fsi, const_cast<float*>(input_k), idx);
}

void sin_cos_cache_calc_cu(int head_size, int max_seq_len,
                           const tensor::Tensor& sin_cache,
                           const tensor::Tensor& cos_cache,
                           hipStream_t stream) {
  CHECK_EQ(sin_cache.is_empty(), false);
  CHECK_EQ(cos_cache.is_empty(), false);
  int threads = head_size;
  if (stream) {
    sin_cos_calc<<<1, threads, 0, stream>>>(
      head_size, max_seq_len,
      const_cast<float*>(sin_cache.ptr<float>()),
      const_cast<float*>(cos_cache.ptr<float>()));
  } else {
    sin_cos_calc<<<1, threads>>>(head_size, max_seq_len,
                                 const_cast<float*>(sin_cache.ptr<float>()),
                                 const_cast<float*>(cos_cache.ptr<float>()));
  }
}

void rope_kernel_cu(int32_t dim, int32_t kv_dim, int32_t head_size,
                    const tensor::Tensor& input_q,
                    const tensor::Tensor& input_k,
                    const tensor::Tensor& input_pos,
                    const tensor::Tensor& sin_cache,
                    const tensor::Tensor& cos_cache, void* stream) {
  const int32_t pos = *input_pos.ptr<int32_t>(0);
  int threads = 128;
  int blocks = (dim + threads - 1) / threads;
  if (stream) {
    hipStream_t stream_ = static_cast<hipStream_t>(stream);
    rope_kernel_cu_fp32<<<blocks, threads, 0, stream_>>>(
      pos, dim, kv_dim, head_size,
      input_q.ptr<float>(), input_k.ptr<float>(),
      sin_cache.ptr<float>(), cos_cache.ptr<float>());
  } else {
    rope_kernel_cu_fp32<<<blocks, threads>>>(
      pos, dim, kv_dim, head_size,
      input_q.ptr<float>(), input_k.ptr<float>(),
      sin_cache.ptr<float>(), cos_cache.ptr<float>());
  }
}
} // namespace kernel
